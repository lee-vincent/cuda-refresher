
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <hip/hip_runtime_api.h>


// Kernel function to add the elements of two arrays
__global__ void add(int n, float *x, float *y)
{
    int index = threadIdx.x;
    int stride = blockDim.x;
    for (int i = index; i < n; i += stride)
        y[i] = x[i] + y[i];
}

int main(void)
{
    int N = 1 << 20;
    std::cout << "N: " << N << std::endl;
    int blockSize = 256;
    std::cout << "blockSize: " << blockSize << std::endl;
    int numBlocks = (N + blockSize - 1) / blockSize;
    std::cout << "numBlocks: " << numBlocks << std::endl;

    float *x, *y;

    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&x, N * sizeof(float));
    hipMallocManaged(&y, N * sizeof(float));

    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++)
    {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    hipProfilerStart();
    add<<<1, 256>>>(N, x, y);
    hipDeviceSynchronize();
    hipProfilerStop();

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i] - 3.0f));
    std::cout << "Max error: " << maxError << std::endl;

    // Free memory
    hipFree(x);
    hipFree(y);

    return 0;
}